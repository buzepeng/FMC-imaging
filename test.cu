#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
int main(){
    int data[6] = {1, 0, 2, 2, 1, 3};
    int *d_data;
    hipMalloc((void **)&d_data, 6*sizeof(int));
    hipMemcpy(d_data, data, 6*sizeof(int), hipMemcpyHostToDevice);
    int h_result, *d_result = thrust::max_element(thrust::device, d_data, d_data + 6);
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    std::cout<<h_result<<std::endl;
    hipFree(d_data);
}