#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>  
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h> 
#include <>
#include <hip/device_functions.h>
#define NX 1500
#define NY 1000
#define NXY NX*NY
#define TILEW 16
#define TILEH 16

using namespace std;

__global__ void hilbertFreqFilter(hipfftComplex* signal){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int idx = ix + iy * NX;
    if(ix<NX && iy<NY){
        signal[idx].x = ((ix==0)+2*(ix>0 && ix < NX/2)+0*(ix>=NX/2 && ix < NX))*signal[idx].x;      //对判断条件进行代数运算，避免分支
        signal[idx].y = ((ix==0)+2*(ix>0 && ix < NX/2)+0*(ix>=NX/2 && ix < NX))*signal[idx].y;
        __syncthreads();
    }
}

__global__ void computEnvelope(float* origin, hipfftComplex* filtered){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int idx = ix + iy * NX;

    if(ix<NX && iy<NY){
        float yi = origin[idx] - filtered[idx].x/NX;    //cufft没有考虑信号的尺度变换
        origin[idx] = abs(yi);
        __syncthreads();
    }
}

cv::Mat hilbert(   float* signals,
                hipfftReal* d_inputData,
                hipfftComplex* d_fftOutput,
                hipfftHandle planForward,
                hipfftHandle planInverse
){
    //将数据拷贝到显存，执行FFT正变换
    hipMemcpy(d_inputData, (hipfftReal*)signals, NXY * sizeof(hipfftReal), hipMemcpyHostToDevice);
    hipfftExecR2C(planForward, d_inputData, d_fftOutput);
    dim3 block(TILEH, TILEW, 1);
    dim3 grid((NX-block.x)/block.x+1, (NY-block.y)/block.y+1, 1);
    //希尔伯特频域滤波
    hilbertFreqFilter<<<grid, block>>>(d_fftOutput);
    hipDeviceSynchronize();
    //执行FFT反变换
    hipfftExecC2C(planInverse, d_fftOutput, d_fftOutput, HIPFFT_BACKWARD);
    hipDeviceSynchronize();
    //计算信号包络
    computEnvelope<<<grid, block>>>(d_inputData, d_fftOutput);
    hipDeviceSynchronize();
    //将显存中的数据拷贝至内存，并进行设备主机同步
    cv::Mat result = cv::Mat::zeros(NY, NX, CV_32FC1);
    hipMemcpy(signals, d_inputData, sizeof(float)*NXY, hipMemcpyDeviceToHost);
    memcpy(result.ptr<float>(0), signals, sizeof(float)*NXY);
    hipDeviceSynchronize();
    return result;
}

int main(){
    //申请锁页内存和显存
    float *input;
    hipHostMalloc((void**)&input, (size_t)sizeof(float)*NXY);
    hipfftReal *d_inputData;
	hipfftComplex *d_fftOutput;
	hipMalloc((void**) &d_inputData, NXY * sizeof(hipfftReal));
	hipMalloc((void**) &d_fftOutput, NXY * sizeof(hipfftComplex));
    //打开文件，将txt中的数据读取到input数组中
    ifstream fp_input;
	ofstream fp_output;
    fp_input.open("./data/data.txt", ios::in);
	if (!fp_input) { //打开失败
        cout << "error opening source file." << endl;
        return 0;
    }
	fp_output.open("./data/output_gpu.txt", ios::out);
	if (!fp_output) {
        fp_input.close(); //程序结束前不能忘记关闭以前打开过的文件
        cout << "error opening destination file." << endl;
        return 0;
    }
    for(int i = 0;i<NY;i++){
        for(int j = 0;j<NX; j++){
            fp_input>>input[i*NX+j];
        }
    }
    //定义两次FFT的类型
    hipfftHandle planForward, planInverse;
    int rank=1;
	int n[1];
	n[0]=NX;
	int istride=1;
	int idist = NX;
	int ostride=1;
	int odist = NX;
	int inembed[2];
	int onembed[2];
	inembed[0]=NX;  onembed[0]=NX;
	inembed[1] = NY; onembed[1] = NY;
    hipfftPlanMany(&planForward,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_R2C, NY);
    hipfftPlanMany(&planInverse,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_C2C, NY);
    //进行希尔伯特变换并计时
    auto startTime = chrono::system_clock::now();
    cv::Mat result = hilbert(input, d_inputData, d_fftOutput, planForward, planInverse);
	auto endTime = chrono::system_clock::now();
    cout << "gpu time:" << chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count() <<"ms"<< endl;
    for(int i = 0;i<NY;i++){
        for(int j = 0;j<NX;j++){
            // if(j!=NX-1) fp_output<<input[i*NX+j]<<'\t';
            // else fp_output<<input[i*NX+j];
            if(j!=NX-1) fp_output<<result.at<float>(i,j)<<'\t';
            else fp_output<<result.at<float>(i,j);
        }
        fp_output<<'\n';
    }
	//释放内存，关闭文件
	fp_input.close();
	fp_output.close();
    hipfftDestroy(planForward);
    hipfftDestroy(planInverse);
    hipHostFree(input);
    hipFree(d_inputData);
    hipFree(d_fftOutput);
	return 0;
}