#include <iostream>
#include <stdio.h>
#include <fstream>  
#include <vector>
#include <math.h>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h> 
#include <chrono>

#define NX 4000
#define NY 4096
#define TILEW 16
#define TILEH 16
#define TAPS 128

__constant__ hipfftComplex H[NX];

__device__ hipfftComplex operator * (hipfftComplex a, hipfftComplex b){
    hipfftComplex res;
    res.x = (a.x*b.x - a.y*b.y);
    res.y = (a.x*b.y + a.y*b.x);
    return res;
}

__global__ void FreqDomainMul(hipfftComplex* signals){
    int ix = blockDim.x*blockIdx.x+threadIdx.x;
    int iy = blockDim.y*blockIdx.y+threadIdx.y;
    int idx = iy*NX+ix;
    int tile_idx = threadIdx.y*TILEW+threadIdx.x;

    __shared__ hipfftComplex filter_s[TILEW], signals_s[TILEW*TILEH];
    filter_s[threadIdx.x] = H[ix];
    signals_s[tile_idx] = signals[idx];
    __syncthreads();

    signals_s[tile_idx] = signals_s[tile_idx] * filter_s[threadIdx.x];
    signals[idx] = signals_s[tile_idx];
    __syncthreads();
}

__global__ void Real2Complex(hipfftReal* in, hipfftComplex* out){
    int ix = blockDim.x*blockIdx.x+threadIdx.x;
    int iy = blockDim.y*blockIdx.y+threadIdx.y;
    int idx = iy*NX+ix;

    out[idx].x = in[idx];
    out[idx].y = 0;
    __syncthreads();
}

__global__ void normalization(hipfftReal* in){
    int ix = blockDim.x*blockIdx.x+threadIdx.x;
    int iy = blockDim.y*blockIdx.y+threadIdx.y;
    int idx = iy*NX+ix;

    in[idx] = in[idx]/(2*NX);
    __syncthreads();
}

__global__ void warmup(){

}

static float sinc(const float x)
{
    if (x == 0)
        return 1;

    return sin(M_PI * x) / (M_PI * x);
}

class HilbertBandPassFilter{
    private:
        float *input, *h;
        hipfftReal *d_inputData, *d_h_in;
        hipfftComplex *d_fftOutput, *d_h_out;
        hipfftHandle planForward, planInverse, planFilter;
    public:
        HilbertBandPassFilter(float f1, float f2){

            // cudaFree(0);
            warmup<<<1,1>>>();
            hipHostMalloc((void**)&input, (size_t)sizeof(float)*NX*NY, hipHostMallocDefault);
            hipHostMalloc((void**)&h, (size_t)sizeof(float)*NX, hipHostMallocDefault);
            hipMalloc((void**) &d_h_in, NX * sizeof(hipfftReal));
	        hipMalloc((void**) &d_h_out, NX * sizeof(hipfftComplex));
            hipMalloc((void**) &d_inputData, NX * NY * sizeof(hipfftReal));
	        hipMalloc((void**) &d_fftOutput, NX * NY * sizeof(hipfftComplex));

            // int rank=1, istride=1, idist = NX, ostride=1, odist = NX;
            // int n[1] = {NX}, inembed[2] = {NY, NX}, onembed[2] = {NY, NX};
            int rank=1;
            int n[1];
            n[0]=NX;
            int istride=1;
            int idist = NX;
            int ostride=1;
            int odist = NX;
            int inembed[2];
            int onembed[2];
            inembed[0]=NX;  onembed[0]=NX;
            inembed[1] = NY; onembed[0] = NY;

            // cufftPlanMany(&planForward,rank,n,inembed, istride ,idist , onembed, ostride,odist, CUFFT_R2C, NY);
            hipfftPlanMany(&planForward,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_R2C, NY);
            hipfftPlanMany(&planInverse,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_C2R, NY);
            hipfftPlan1d(&planFilter, NX, HIPFFT_R2C, 1);

            //generate bandpas filter
            memset(h, 0, NX);
            for(int i = 0; i < TAPS; i++) {
                int n = i - int(TAPS/2);
                float w =  sin(((float) M_PI * i) / (TAPS - 1)) *
                        sin(((float) M_PI * i) / (TAPS - 1));
                h[i] = 2.0*f1*sinc(2.0*f1*n) - 2.0*f2*sinc(2.0*f2*n);
                h[i] = w * h[i];
            }

            hipMemcpy(d_h_in, (hipfftReal*)h, NX * sizeof(hipfftReal), hipMemcpyHostToDevice);
            hipfftExecR2C(planFilter, d_h_in, d_h_out);
            hipMemcpyToSymbol( HIP_SYMBOL(H), d_h_out, sizeof(hipfftComplex)*NX, 0, hipMemcpyDeviceToDevice );

            //generate hilbert filter
            for(int i = 0;i<NX;i++){
                if(i == 0 || (float)i == NX/2.0){
                    h[i] = 1;
                }else if(i<NX/2){
                    h[i] = 2;
                }else{
                    h[i] = 0;
                }
            }
            hipMemcpy(d_h_in, (hipfftReal*)h, NX * sizeof(hipfftReal), hipMemcpyHostToDevice);
            dim3 block(TILEW, 1, 1);
            dim3 grid(ceil(NX/block.x), 1, 1);
            Real2Complex<<<grid, block>>>(d_h_in, d_h_out);
            FreqDomainMul<<<grid, block>>>(d_h_out);
            hipMemcpyToSymbol( HIP_SYMBOL(H), d_h_out, sizeof(hipfftComplex)*NX, 0, hipMemcpyDeviceToDevice );
        }
        float* read_data(std::string file_path){
            std::ifstream fp_input;
            fp_input.open(file_path, std::ios::in);
            if (!fp_input) { //打开失败
                std::cout << "error opening source file." << std::endl;
                return nullptr;
            }
            for(int i = 0;i<NY;i++){
                for(int j = 0;j<NX; j++){
                    fp_input>>input[i*NX+j];
                }
            }
            fp_input.close();
            return input;
        }

        float* filter(float* signals){
            hipMemcpy(d_inputData, (hipfftReal*)signals, NX * NY * sizeof(hipfftReal), hipMemcpyHostToDevice);
            hipfftExecR2C(planForward, d_inputData, d_fftOutput);
            dim3 block(TILEW, TILEH, 1);
            dim3 grid(ceil(NX/block.x), ceil(NY/block.y), 1);
            FreqDomainMul<<<grid, block>>>(d_fftOutput);
            //执行FFT反变换
            hipfftExecC2R(planInverse, d_fftOutput, d_inputData);
            normalization<<<grid, block>>>(d_inputData);
            hipMemcpy(input, (float*)d_inputData, (size_t)sizeof(float)*NX*NY, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            return input;
        }

        void save_result(float* result, std::string filepath){
            std::ofstream fp_output;
            fp_output.open(filepath, std::ios::out);
            if (!fp_output) {
                std::cout << "error opening destination file." << std::endl;
                return;
            }
            for(int i = 0;i<NY;i++){
                for(int j = 0;j<NX;j++){
                    // std::cout<<result[i*NX+j]<<" "<<std::endl;
                    if(j!=NX-1){
                        fp_output<<result[i*NX+j]<<' ';
                    }else{
                        fp_output<<result[i*NX+j]<<'\n';
                    } 
                }
            }
            fp_output.close();
        }
        ~HilbertBandPassFilter(){
            hipfftDestroy(planForward);
            hipfftDestroy(planInverse);
            hipfftDestroy(planFilter);
            hipHostFree(input);
            hipHostFree(h);
            hipFree(d_inputData);
            hipFree(d_fftOutput);
            hipFree(d_h_out);
            hipFree(d_h_in);
        }
};


int main(){
    float *signals, *results;
    float fs = 1000, fl = 15.0/fs, fh = 45.0/fs;

    HilbertBandPassFilter hf(fl, fh);
    
    signals = hf.read_data("/home/hilbert/data/input_data.txt");
    std::cout<<"Signal read complete!"<<std::endl;

    auto start = std::chrono::high_resolution_clock::now();
    results = hf.filter(signals);
    auto end = std::chrono::high_resolution_clock::now();
    std::cout<<"Result compute complete! Time spend: " << std::chrono::duration<float>(end - start).count()<<std::endl;
    hf.save_result(results, "/home/hilbert/data/output_data.txt");

	return 0;
}