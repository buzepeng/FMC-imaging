#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>  
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h> 
#include <>
#define NX 1500
#define NY 1000
#define NXY NX*NY
#define TILEW 16
#define TILEH 16
#define BATCH 64

using namespace std;

__global__ void hilbertFreqFilter(hipfftComplex* signal){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int iz = blockIdx.z;
    int idx = ix + iy * NX + iz*NXY;
    if(ix<NX && iy<NY){
        signal[idx].x = ((ix==0)+2*(ix>0 && ix < NX/2)+0*(ix>=NX/2 && ix < NX))*signal[idx].x;
        signal[idx].y = ((ix==0)+2*(ix>0 && ix < NX/2)+0*(ix>=NX/2 && ix < NX))*signal[idx].y;
        __syncthreads();
    }
}

__global__ void computEnvelope(float* origin, hipfftComplex* filtered){
    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int iz = blockIdx.z;
    int idx = ix + iy * NX + iz*NXY;

    if(ix<NX && iy<NY){
        // float xr = origin[idx];
        // float yr = filtered[idx].y;
        // float yi = xr - filtered[idx].x;
        float yi = origin[idx] - filtered[idx].x/NX;
        // float amp = sqrtf(yr*yr*yi*yi);
        origin[idx] = abs(yi);
        __syncthreads();
    }
}

void hilbert(float* signals){

    hipfftReal *d_inputData;
	hipfftComplex *d_fftOutput;
	hipMalloc((void**) &d_inputData, BATCH*NXY * sizeof(hipfftReal));
	hipMalloc((void**) &d_fftOutput, BATCH*NXY * sizeof(hipfftComplex));
	hipMemcpy(d_inputData, (hipfftReal*)signals, BATCH*NXY * sizeof(hipfftReal), hipMemcpyHostToDevice);
	hipfftHandle plan;

    int rank=1;
	int n[1];
	n[0]=NX;
	int istride=1;
	int idist = NX;
	int ostride=1;
	int odist = NX;
	int inembed[3];
	int onembed[3];
	inembed[0]=NX;  onembed[0]=NX;
	inembed[1] = NY; onembed[1] = NY;
    inembed[2] = BATCH; onembed[2] = BATCH;

    hipfftPlanMany(&plan,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_R2C, NY*BATCH);
    hipfftExecR2C(plan, d_inputData, d_fftOutput);
    dim3 block(TILEH, TILEW, 1);
    dim3 grid((NX-block.x)/block.x+1, (NY-block.y)/block.y+1, BATCH);
    hilbertFreqFilter<<<grid, block>>>(d_fftOutput);
    hipDeviceSynchronize();

    hipfftPlanMany(&plan,rank,n,inembed, istride ,idist , onembed, ostride,odist, HIPFFT_C2C, NY*BATCH);
    hipfftExecC2C(plan, d_fftOutput, d_fftOutput, HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    computEnvelope<<<grid, block>>>(d_inputData, d_fftOutput);
    hipDeviceSynchronize();

    hipMemcpy(signals, d_inputData, sizeof(float)*BATCH*NXY, hipMemcpyDeviceToHost);
    hipFree(d_inputData);
    hipFree(d_fftOutput);
}

int main(){
    ifstream fp_input;
    fp_input.open("./data/data.txt", ios::in);
	if (!fp_input) { //打开失败
        cout << "error opening source file." << endl;
        return 0;
    }
    cv::Mat src_data = cv::Mat::zeros(NY, NX, CV_32FC1);
    for(int i = 0;i<NY;i++){
        for(int j = 0;j<NX; j++){
            fp_input >> src_data.at<float>(i, j);
        }
    }
    // float *input = new float[NXY]();
    float *input;
    hipHostMalloc((void**)&input, (size_t)sizeof(float)*NXY*BATCH);
    for(int i = 0;i<BATCH;i++){
        memcpy(input+i*NXY, src_data.ptr<float>(0, 0), sizeof(float)*NXY);
    }
    auto startTime = chrono::system_clock::now();
    hilbert(input);
	auto endTime = chrono::system_clock::now();
	cout << "gpu time:" << chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count()<<"ms"<< endl;
	
	fp_input.close();
    hipHostFree(input);
	return 0;
}